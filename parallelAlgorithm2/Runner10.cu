#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

const unsigned int N = 1 << 20;
const unsigned int MEMSIZE = N * sizeof(unsigned int);
const unsigned int NO_LOOPS = 100;
const unsigned int THREAD_PER_BLOCK = 256;
const unsigned int GRID_SIZE = (N + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK;

void fillData(unsigned int *data, const unsigned int length)
{
	//srand(time(0));
	for (unsigned int i=0; i<length; i++)
	{
		//data[i]= rand();
		data[i]= 1;
	}
}

void printData(const unsigned int *data, const unsigned int length)
{
	if (data ==0) return;
	for (unsigned int i=0; i<length; i++)
	{
		printf("%u ", data[i]);
	}
}


__global__ void kernel(const unsigned int *a, const unsigned int *b, const unsigned int length, unsigned int *c)
{
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//TODO:  thread block loop
	if (tid < length)
	{
		c[tid] = a[tid] + b[tid];
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1()
{
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipHostAlloc( (void**)&da, MEMSIZE, hipHostMallocDefault);
	hipHostAlloc( (void**)&db, MEMSIZE, hipHostMallocDefault);
	hipHostAlloc( (void**)&dc, MEMSIZE, hipHostMallocDefault);

	hipStream_t stream;
	hipStreamCreate(&stream);

	unsigned int dataOffset = 0;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for(int i=0; i < NO_LOOPS; i++)
	{
		

		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream);

		kernel <<< GRID_SIZE, THREAD_PER_BLOCK,0,stream>>>(da, db, N, dc);
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyHostToDevice, stream);
		dataOffset += N;
	}

	//TODO: Synchonize stream
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test time: %f ms\n", elapsedTime);

	printData(c, 100);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2()
{
	unsigned int *a, *b, *c;
	unsigned int *da1, *db1, *dc1, *da2, *db2, *dc2;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da1, MEMSIZE);
	hipMalloc((void**)&db1, MEMSIZE);
	hipMalloc((void**)&dc1, MEMSIZE);

	hipMalloc((void**)&da2, MEMSIZE);
	hipMalloc((void**)&db2, MEMSIZE);
	hipMalloc((void**)&dc2, MEMSIZE);

	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	unsigned int dataOffset = 0;

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	for (int i = 0; i < NO_LOOPS; i += 2)
	{

		hipMemcpyAsync(da1, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db1, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1);
		kernel << < GRID_SIZE, THREAD_PER_BLOCK, 0, stream1 >> >(da1, db1, N, dc1);
		hipMemcpyAsync(&c[dataOffset], dc1, MEMSIZE, hipMemcpyHostToDevice, stream1);

		dataOffset += N;

		hipMemcpyAsync(da2, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(db2, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream2);
		kernel << < GRID_SIZE, THREAD_PER_BLOCK, 0, stream2 >> >(da1, db1, N, dc1);
		hipMemcpyAsync(&c[dataOffset], dc2, MEMSIZE, hipMemcpyHostToDevice, stream2);
		
		dataOffset += N;
	}

	//TODO: Synchonize stream
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test time: %f ms\n", elapsedTime);

	printData(c, 100);

	hipFree(da1);
	hipFree(db1);
	hipFree(dc1);

	hipFree(da2);
	hipFree(db2);
	hipFree(dc2);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3()
{
	//TODO: reuse the source code of above mentioned method test1()
}


int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	test1();
	test2();
	test3();

	return 0;
}
