#include "hip/hip_runtime.h"
#include "Runner3.cuh"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <random>
#include <hip/device_functions.h>

//WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
constexpr unsigned int TPB = 128;
constexpr unsigned int NO_FORCES = 256;
constexpr unsigned int NO_RAIN_DROPS = 1 << 20;

constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using namespace std;

__host__ float3 *createData(const unsigned int length)
{
	random_device rd;
	mt19937_64 mt(rd());
	uniform_int_distribution<float> dist(0.0f, 1.0f);

	float3 *data = static_cast<float3*>(::operator new(sizeof(float3)* length));

	for (unsigned int i = 0; i < length; i++) {
		//data[i] = make_float3(dist(mt), dist(mt), dist(mt));
		data[i] = make_float3(1.0f, 1.0f, 1.0f);
	}

	//TODO: Generate float3 vectors. You can use 'make_float3' method.
	return data;
}

void printData(const float3 *data, const unsigned int length)
{
	if (data == 0) return;
	const float3 *ptr = data;
	for (unsigned int i = 0; i<length; i++, ptr++)
	{
		printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Sums the forces to get the final one using parallel reduction. 
/// 		    WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
/// <param name="dForces">	  	The forces. </param>
/// <param name="noForces">   	The number of forces. </param>
/// <param name="dFinalForce">	[in,out] If non-null, the final force. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
extern void reduce(const float3 * __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce)
{
	
	
	__shared__ float3 sForces[TPB];		//SEE THE WARNING MESSAGE !!!
	unsigned int tid = threadIdx.x;
	unsigned int next = TPB;			//SEE THE WARNING MESSAGE !!!
	
	float3 * src = &sForces[tid];
	float3 * src2 = (float3*)&sForces[tid + next];

	#pragma unroll
	for (unsigned int s = (blockDim.x >> 1); s>32; s >>= 1)
	{
		if (tid >= s) return;

		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;
		src2 = src + s;
		__syncthreads();
	}

	volatile float3 *vsrc = &sForces[tid];

	if (tid < 32)
	{
		vsrc[tid].x += vsrc[tid + 32].x;
		vsrc[tid].y += vsrc[tid + 32].y;
		vsrc[tid].z += vsrc[tid + 32].z;

		vsrc[tid].x += vsrc[tid + 16].x;
		vsrc[tid].y += vsrc[tid + 16].y;
		vsrc[tid].z += vsrc[tid + 16].z;

		vsrc[tid].x += vsrc[tid + 8].x;
		vsrc[tid].y += vsrc[tid + 8].y;
		vsrc[tid].z += vsrc[tid + 8].z;

		vsrc[tid].x += vsrc[tid + 4].x;
		vsrc[tid].y += vsrc[tid + 4].y;
		vsrc[tid].z += vsrc[tid + 4].z;

		vsrc[tid].x += vsrc[tid + 2].x;
		vsrc[tid].y += vsrc[tid + 2].y;
		vsrc[tid].z += vsrc[tid + 2].z;

		vsrc[tid].x += vsrc[tid + 1].x;
		vsrc[tid].y += vsrc[tid + 1].y;
		vsrc[tid].z += vsrc[tid + 1].z;
	}

	dFinalForce->x = vsrc->x;
	dFinalForce->y = vsrc->y;
	dFinalForce->z = vsrc->z;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds the FinalForce to every Rain drops position. </summary>
/// <param name="dFinalForce">	The final force. </param>
/// <param name="noRainDrops">	The number of rain drops. </param>
/// <param name="dRainDrops"> 	[in,out] If non-null, the rain drops positions. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops)
{
	//TODO: Add the FinalForce to every Rain drops position.
}


int particleSystemSimulation()
{
	initializeCUDA(deviceProp);

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	float3 *hForces = createData(NO_FORCES);
	float3 *hDrops = createData(NO_RAIN_DROPS);

	float3 *dForces = nullptr;
	float3 *dDrops = nullptr;
	float3 *dFinalForce = nullptr;

	error = hipMalloc((void**)&dForces, NO_FORCES * sizeof(float3));
	error = hipMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), hipMemcpyHostToDevice);

	error = hipMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3));
	error = hipMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), hipMemcpyHostToDevice);

	error = hipMalloc((void**)&dFinalForce, sizeof(float3));

	KernelSetting ksReduce;

	//TODO: ... Set ksReduce


	KernelSetting ksAdd;
	//TODO: ... Set ksAdd
	
	for (unsigned int i = 0; i<1000; i++)
	{
		reduce<<<ksReduce.dimGrid, ksReduce.dimBlock>>>(dForces, NO_FORCES, dFinalForce);
		add<<<ksAdd.dimGrid, ksAdd.dimBlock>>>(dFinalForce, NO_RAIN_DROPS, dDrops);
	}

	checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
	// checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");

	if (hForces)
		free(hForces);
	if (hDrops)
		free(hDrops);

	hipFree(dForces);
	hipFree(dDrops);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	printf("Time to get device properties: %f ms", elapsedTime);
}
