#include "hip/hip_runtime.h"
// includes, cuda
#include <hip/hip_runtime.h>

#include <cudaDefs.h>
#include <imageManager.h>


#include "imageKernels.cuh"

#define BLOCK_DIM 8

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

texture<float, hipTextureType2D, hipReadModeElementType> texRef;
hipChannelFormatDesc texChannelDesc;

unsigned char *dImageData = 0;
unsigned int imageWidth;
unsigned int imageHeight;
unsigned int imageBPP;		//Bits Per Pixel = 8, 16, 24, or 32 bit
unsigned int imagePitch;

size_t texPitch;
float *dLinearPitchTextureData = 0;
hipArray *dArrayTextureData = 0;

uchar3 *dstTexData;

KernelSetting squareKs;

float *dOutputData = 0;

__constant__  int SOBEL_X_FILTER[] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
__constant__  int SOBEL_Y_FILTER[] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };

template<bool normalizeTexel>__global__ void floatHeighmapTextureToNormalmap(const unsigned int texWidth, const unsigned int texHeight, const unsigned int dstPitch, uchar3* dst)
{

	unsigned int col = (threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int row = (threadIdx.y + blockIdx.y * blockDim.y);

	float3 floatTexel;
	floatTexel.z = 0.5;
	unsigned int offset = col + row * (dstPitch / 3);
	
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			float texel = tex2D(texRef, col + (j - 1), row + (i - 1));
			floatTexel.x += texel * SOBEL_X_FILTER[j + i * 3];
			floatTexel.y += texel * SOBEL_Y_FILTER[j + i * 3];
		}
	}
 
	floatTexel = normalize(floatTexel);

	uchar3 rgbTexel;
	uchar3 bgrTexel;
	rgbTexel.x = (floatTexel.x + 1) * 127.5;
	rgbTexel.y = (floatTexel.y + 1) * 127.5;
	rgbTexel.z = floatTexel.z * 255;

	bgrTexel.x = rgbTexel.z;
	bgrTexel.y = rgbTexel.y;
	bgrTexel.z = rgbTexel.x;
	dst[offset] = rgbTexel;
}

#pragma region STEP 1

//TASK:	Load the input image and store loaded data in DEVICE memory (dSrcImageData)

void loadSourceImage(const char* imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);

	imageWidth = FreeImage_GetWidth(tmp);
	imageHeight = FreeImage_GetHeight(tmp);
	imageBPP = FreeImage_GetBPP(tmp);
	imagePitch = FreeImage_GetPitch(tmp);		// FREEIMAGE aligns row data ... You have to use pitch instead of width

	hipMalloc((void**)&dImageData, imagePitch * imageHeight * imageBPP / 8);
	hipMemcpy(dImageData, FreeImage_GetBits(tmp), imagePitch * imageHeight * imageBPP / 8, hipMemcpyHostToDevice);

	//checkHostMatrix<unsigned char>(FreeImage_GetBits(tmp), imagePitch, imageHeight, imageWidth, "");
	//checkDeviceMatrix<unsigned char>(dImageData, imagePitch, imageHeight, imageWidth, "", "");

	FreeImage_Unload(tmp);
	FreeImage_DeInitialise();
}
#pragma endregion

#pragma region STEP 2

//TASK: Create a texture based on the source image. The input images can have variable BPP (Byte Per Pixel), but finally any such image will be converted into the floating-point texture using
//		the colorToFloat kernel.

void createSrcTexure()
{
	//Floating Point Texture Data
	hipMallocPitch((void**)&dLinearPitchTextureData, &texPitch, imageWidth * sizeof(float), imageHeight);

	//Converts custom image data to float and stores result in the float_pitch_linear_data
	switch (imageBPP)
	{
	case 8:  colorToFloat<8, 2> << <squareKs.dimGrid, squareKs.dimBlock >> > (dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData); break;
	case 16: colorToFloat<16, 2> << <squareKs.dimGrid, squareKs.dimBlock >> > (dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData); break;
	case 24: colorToFloat<24, 2> << <squareKs.dimGrid, squareKs.dimBlock >> > (dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData); break;
	case 32: colorToFloat<32, 2> << <squareKs.dimGrid, squareKs.dimBlock >> > (dImageData, imageWidth, imageHeight, imagePitch, texPitch / sizeof(float), dLinearPitchTextureData); break;
	}

	//checkDeviceMatrix<float>(dLinearPitchTextureData, texPitch, imageHeight, imageWidth, "", "");

	//Texture settings
	texChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	texRef.normalized = false;
	texRef.filterMode = hipFilterModePoint;
	texRef.addressMode[0] = hipAddressModeWrap;
	texRef.addressMode[1] = hipAddressModeWrap;

	hipBindTexture2D(0, &texRef, dLinearPitchTextureData, &texChannelDesc, imageWidth, imageHeight, texPitch);
}
#pragma endregion

#pragma region STEP 3

//TASK:	Convert the input image into normal map. Use the binded texture (srcTexRef).

void createNormalMap()
{
	checkCudaErrors(hipMallocPitch((void**)&dstTexData, &texPitch, imageWidth * 3, imageHeight));
	floatHeighmapTextureToNormalmap<true> << <squareKs.dimGrid, squareKs.dimBlock>> >(imageWidth, imageHeight, texPitch, dstTexData);
	//check_data<uchar3>::checkDeviceMatrix(dstTexData, imageHeight, texPitch / sizeof(uchar3), true, "%hhu %hhu %hhu | ", "Result of Linear Pitch Text");
}

#pragma endregion

#pragma region STEP 4

//TASK: Save output image (normal map)

void saveTexImage(const char* imageFileName)
{
	FreeImage_Initialise();

	FIBITMAP *tmp = FreeImage_Allocate(imageWidth, imageHeight, 24);
	unsigned int tmpPitch = imagePitch = FreeImage_GetPitch(tmp);		// FREEIMAGE align row data ... You have to use pitch instead of width
	checkCudaErrors(hipMemcpy2D(FreeImage_GetBits(tmp), FreeImage_GetPitch(tmp), dstTexData, texPitch, imageWidth * 3, imageHeight, hipMemcpyDeviceToHost));
	//FreeImage_Save(FIF_PNG, tmp, imageFileName, 0);
	ImageManager::GenericWriter(tmp, imageFileName, FIF_PNG);
	FreeImage_Unload(tmp);
	FreeImage_DeInitialise();
}

#pragma endregion

void releaseMemory()
{
	hipUnbindTexture(texRef);
	if (dImageData != 0)
		hipFree(dImageData);
	if (dLinearPitchTextureData != 0)
		hipFree(dLinearPitchTextureData);
	if (dArrayTextureData)
		hipFreeArray(dArrayTextureData);
	if (dOutputData)
		hipFree(dOutputData);
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	//STEP 1
	loadSourceImage("C:/Users/kne0035/dev/projects/parallelAlgorithm2/parallelAlgorithm2/images/terrain3Kx3K.tif");

	//TODO: Setup the kernel settings
	squareKs.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	squareKs.blockSize = BLOCK_DIM * BLOCK_DIM;
	squareKs.dimGrid = dim3((imageWidth + BLOCK_DIM - 1) / BLOCK_DIM, (imageHeight + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	//Step 2 - create heighmap texture stored in the linear pitch memory
	createSrcTexure();

	//Step 3 - create the normal map
	createNormalMap();

	//Step 4 - save the normal map
	saveTexImage("C:/Users/kne0035/dev/projects/parallelAlgorithm2/parallelAlgorithm2/images/normalMap.bmp");

	releaseMemory();
}
