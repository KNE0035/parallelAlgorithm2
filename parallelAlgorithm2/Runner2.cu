#include <cudaDefs.h>


hipDeviceProp_t deviceProp = hipDeviceProp_t();

void createMatrixOnDevice(unsigned int mRows, unsigned int mCols);


int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);
	
	
	
	
	testAdding2Vectors();
	testAddingNVectors();
}


void createMatrixOnDevice(unsigned int mRows, unsigned int mCols) {
	unsigned int* dMatrix = new unsigned int[mRows * mCols];
	unsigned int* pitch;


	hipMallocPitch((void**)&dMatrix, &pitch, )



}