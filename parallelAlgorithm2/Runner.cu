#include "hip/hip_runtime.h"
#include <cudaDefs.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

void testAdding2Vectors();
void testAddingNVectors();


__global__ void add2VectorsKernel(double *resultVector, const double *vectorA, const double *vectorB)
{
	int i = threadIdx.x;
	resultVector[i] = vectorA[i] + vectorB[i];
}

__global__ void addNVectorsKernel(double *resultVector, const double *vectorsArray, const int* n)
{
	int i = threadIdx.x;

	for (int j = 0; j < (*n); j++) {
		resultVector[i] += vectorsArray[i + j * (*n)];
	}
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);
	testAdding2Vectors();
	testAddingNVectors();
}

void testAdding2Vectors () {
	const int dimmensions = 3;

	const double vectorA[dimmensions] = { 2, 2, 3 };
	const double vectorB[dimmensions] = { 4, 5, 6 };
	double resultVector[dimmensions] = { 0 };

	double* devA = 0;
	double* devB = 0;
	double* devResult = 0;

	error = hipMalloc((void**)&devA, dimmensions * sizeof(double));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	error = hipMalloc((void**)&devB, dimmensions * sizeof(double));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	error = hipMalloc((void**)&devResult, dimmensions * sizeof(double));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}


	error = hipMemcpy(devA, vectorA, dimmensions * sizeof(double), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	error = hipMemcpy(devB, vectorB, dimmensions * sizeof(double), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	add2VectorsKernel <<<1, dimmensions >>> (devResult, devA, devB);

	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(error));
	}

	error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", error);
	}

	error = hipMemcpy(resultVector, devResult, dimmensions * sizeof(double), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	printf("Result vector: ");

	for (int i = 0; i < dimmensions; i++) {
		printf(" %f, ", resultVector[i]);
	}

	hipFree(devA);
	hipFree(devB);
	hipFree(devResult);
}

void testAddingNVectors() {
	double* devVectors = 0;
	double* devResult = 0;
	int* devNumberOfVectors = 0;
	
	const int dimmensions = 3;
	const int numberOfVectors = 3;

	double resultVector[dimmensions] = { 0 };
	double* vectorsArray = new double[dimmensions * numberOfVectors];
	std::fill(vectorsArray, vectorsArray + (dimmensions * numberOfVectors), 3);

	error = hipMalloc((void**)&devVectors, dimmensions * numberOfVectors * sizeof(double));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	error = hipMalloc((void**)&devNumberOfVectors, sizeof(int));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	error = hipMalloc((void**)&devResult, dimmensions * sizeof(double));
	if (error != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	error = hipMemcpy(devVectors, vectorsArray, dimmensions * numberOfVectors * sizeof(double), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	error = hipMemcpy(devNumberOfVectors, &numberOfVectors, sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	addNVectorsKernel <<<1, dimmensions >>> (devResult, devVectors, devNumberOfVectors);

	error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", error);
	}

	error = hipMemcpy(resultVector, devResult, dimmensions * sizeof(double), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	printf("Result vector: ");

	for (int i = 0; i < dimmensions; i++) {
		printf(" %f, ", resultVector[i]);
	}

	hipFree(devVectors);
	hipFree(devResult);
	hipFree(devNumberOfVectors);
}