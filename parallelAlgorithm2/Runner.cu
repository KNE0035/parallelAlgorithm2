#include "hip/hip_runtime.h"
#include "Runner.cuh"
#include "cudaUtils.cuh"

#include <cudaDefs.h>


hipDeviceProp_t deviceProp = hipDeviceProp_t();


constexpr unsigned int THREADS_PER_BLOCK = 256;

void testAdding2Vectors();
void testAddingNVectors();


__global__ void add2VectorsKernel(float *resultVector, const float *vectorA, const float *vectorB, const unsigned int lenght)
{
	unsigned int offset = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	
	while (offset < lenght) {
		resultVector[offset] = vectorA[offset] + vectorB[offset];
		offset += gridDim.x * THREADS_PER_BLOCK;
	}
}

__global__ void addNVectorsKernel(double *resultVector, const double *vectorsArray, const unsigned int nOVectors, const unsigned int vectorsLength)
{
	unsigned int offset = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	while (offset < vectorsLength) {
		for (unsigned int j = 0; j < nOVectors; j++) {
			resultVector[offset] += vectorsArray[offset + j * nOVectors];
		}
		offset += gridDim.x * THREADS_PER_BLOCK;
	}
}

void testAdding2Vectors () {
	initializeCUDA(deviceProp);
	
	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	const int dimmensions = 10;

	float* vectorA= new float[dimmensions];
	float* vectorB = new float[dimmensions];
	float* resultVector = new float[dimmensions];

	std::fill(vectorA, vectorA + (dimmensions), 1000);
	std::fill(vectorB, vectorB + (dimmensions), 1000);

	float* devA = 0;
	float* devB = 0;
	float* devResult = 0;

	gpuErrorCheck(hipMalloc((void**)&devA, dimmensions * sizeof(float)));
	gpuErrorCheck(hipMalloc((void**)&devB, dimmensions * sizeof(float)));
	gpuErrorCheck(hipMalloc((void**)&devResult, dimmensions * sizeof(float)));

	gpuErrorCheck(hipMemcpy(devA, vectorA, dimmensions * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(devB, vectorB, dimmensions * sizeof(float), hipMemcpyHostToDevice));

	dim3 DimGrid((dimmensions - 1) / THREADS_PER_BLOCK + 1);
	dim3 DimBlock(THREADS_PER_BLOCK);

	hipEventRecord(startEvent, 0);

	add2VectorsKernel <<<DimGrid, DimBlock>>> (devResult, devA, devB, dimmensions);

	hipEventRecord(stopEvent, 0);

	gpuErrorCheck(hipDeviceSynchronize());
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);


	gpuErrorCheck(hipMemcpy(resultVector, devResult, dimmensions * sizeof(float), hipMemcpyDeviceToHost));
	printf("\n time: %f \n\n", elapsedTime);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	/*printf("Result vector: ");
	for (int i = 0; i < dimmensions; i++) {
		printf(" %f, ", resultVector[i]);
	}*/

	delete vectorA;
	delete vectorB;
	delete resultVector;

	hipFree(devA);
	hipFree(devB);
	hipFree(devResult);
}

void testAddingNVectors() {
	double* devVectors = 0;
	double* devResult = 0;

	const int dimmensions = 3;
	const int numberOfVectors = 3;

	double resultVector[dimmensions] = { 0 };
	double* vectorsArray = new double[dimmensions * numberOfVectors];
	std::fill(vectorsArray, vectorsArray + (dimmensions * numberOfVectors), 3);

	gpuErrorCheck(hipMalloc((void**)&devVectors, dimmensions * numberOfVectors * sizeof(double)));
	gpuErrorCheck(hipMalloc((void**)&devResult, dimmensions * sizeof(double)));

	gpuErrorCheck(hipMemcpy(devVectors, vectorsArray, dimmensions * numberOfVectors * sizeof(double), hipMemcpyHostToDevice));

	dim3 DimGrid((dimmensions - 1) / THREADS_PER_BLOCK + 1);
	dim3 DimBlock(THREADS_PER_BLOCK);
	addNVectorsKernel <<<DimGrid, DimBlock>>> (devResult, devVectors, dimmensions, numberOfVectors);

	gpuErrorCheck(hipDeviceSynchronize());

	gpuErrorCheck(hipMemcpy(resultVector, devResult, dimmensions * sizeof(double), hipMemcpyDeviceToHost));

	printf("Result vector: ");
	for (int i = 0; i < dimmensions; i++) {
		printf(" %f, ", resultVector[i]);
	}

	delete vectorsArray;

	hipFree(devVectors);
	hipFree(devResult);
}